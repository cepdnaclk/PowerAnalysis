#include "hip/hip_runtime.h"
/*
	
	The A to Z of Building a Testbed for Power Analysis Attacks
	CUDA C source code for Correlation Power Analysis algorithm 

	This is used to generate data for plotting the variation of correlation coefficient with time
	Saves the all correlation coefficients for all key possibilities along time
	This will generate 4096 files under names subbyte<x>_keyguess<y> where <x> and <y> are integers
	
	This program needs open 4096 files at the same time.
	For that you may need log as root using "sudo su"
	then increase the maximum simultaneously opened file limit by "ulimit -n 5000"
	
    Authors : Hasindu Gamaarachchi, Harsha Ganegoda and Roshan Ragel, 
    Department of Computer Engineering, 
    Faculty of Engineering, University of Peradeniya, 22 Dec 2015
 
    For more information read 
    Hasindu Gamaarachchi, Harsha Ganegoda and Roshan Ragel, 
    "The A to Z of Building a Testbed for Power Analysis Attacks", 
    10th IEEE International Conference on Industrial and Information Systems 2015 (ICIIS)]
 
    Any bugs, issues or suggestions please email to hasindu2008@live.com

*/

#include <stdio.h>
#include "helpers.cuh"
#include "data.cuh"

/*************************************************************** CHANGEBLE PARAMETERS ****************************************************************************/
/* Rather that modifying parameters here, it is recommended to run the provided script. The script will take modifiable parameters as arguments 
and will change this source code using "sed" to generate a temporary modified source code called cpatemp.cu
*/


/* file name for all key-correlation pairs sorted in key order
This file after running would contain all the key bytes sorted in key order with their respective correlation coefficients*/
#define FILEALL "all_key_corr.txt"

/* file name for all key-correlation pairs sorted using correlation coefficient
This file after running would contain all the key bytes and their respective correlation coefficients sorted in descending order of the correlation coefficient*/
#define FILEALLSORTED "all_key_corr_sorted.txt"

/* input file format for the power trace file
put 0 for ASCII format power traces. 
put 1 to save power traces in binary. Not readable directly, but file size is less*/
#define WAVFORMAT 1

/* The number of power traces used for calculation*/
#define SAMPLES 1000

/* The number of sampling points in a single power trace
This value can be found by inspecting the stat.txt generated after collecting a set of power traces*/
#define ALLWAVELEN 100000

/*Global memory on a GPU is limited (RAM as well) and hence if the power traces are large sized, all the things won't fit at once to memory
If you experience a memory allocation failure when running, reduce WAVELENGTH value
This will force the program to read the power traces part by part 
But when using this implementation make sure that ALLWAVELENGTH is divisible by WAVELENGTH 
*/
#define WAVELENGTH 25000

/************************************************************ END OF CHANGEBLE PARAMETERS ****************************************************************************/


//defined for 128 bit AES
#define KEYBYTES 16
#define KEYS 256

//struct used for sorting correlation key pairs
struct key_corr{
	unsigned int key;
	double corr;
};

//hamming weight of a number
__device__ byte hammingweight(byte H){

	// Count the number of set bits
	byte dist=0;
	while(H){
		dist++; 
		H &= H - 1;
	}
	return dist;
}

/********************************************************************** SELECTION FUNCTION ****************************************************************************/
//This will have to be modified if your selection function/intermediate values are different

//find hamming weight for the selection function
__device__ byte hamming(unsigned int *sample, unsigned int i,unsigned int n,unsigned int key) { //n is byteno  i is the sample
    byte inter = (byte)sbox[sample[i*KEYBYTES+n] ^ key];
    byte dist = hammingweight(inter);	  
	return dist;
}
/********************************************************************END SELECTION FUNCTION ****************************************************************************/


//find the correlation values and then the maximum
__global__ void maxCorelationkernel(double *corelation,double *wavestat,double *wavestat2,double *hammingstat,double *allcorelation){
	
	int keyguess=blockDim.y*blockIdx.y+threadIdx.y;
	int keybyte=blockDim.x*blockIdx.x+threadIdx.x;

	if (keybyte<KEYBYTES && keyguess<KEYS ){ 
		
		double sigmaH,sigmaH2,sigmaW=0,sigmaW2=0,sigmaWH=0;	
		sigmaH=hammingstat[KEYBYTES*keyguess+keybyte];
		sigmaH2=hammingstat[KEYS*KEYBYTES+KEYBYTES*keyguess+keybyte];
		double temp_corelation=0;;
		double corelationmax=0;;
		unsigned int j;
		for(j=0;j<WAVELENGTH;j++){
			
			sigmaWH=wavestat2[j*KEYS*KEYBYTES + keyguess*KEYBYTES + keybyte];
			sigmaW=wavestat[j];
			sigmaW2=wavestat[WAVELENGTH+j];

			double numerator=SAMPLES*sigmaWH - sigmaW*sigmaH;
			double denominator=sqrt(SAMPLES*sigmaW2 - sigmaW*sigmaW)*sqrt(SAMPLES*sigmaH2 - sigmaH*sigmaH);
			temp_corelation=-numerator/denominator;
			allcorelation[j*KEYS*KEYBYTES + keyguess*KEYBYTES + keybyte]=temp_corelation;
			
			if(temp_corelation>corelationmax){
				corelationmax=temp_corelation;
			}
		}

		if(corelationmax>corelation[keyguess*KEYBYTES+keybyte]){
			corelation[keyguess*KEYBYTES+keybyte]=corelationmax;
		}
	}
	return;
}

//find power trace statistics such as sigmaW sigmaw^2 etc
__global__ void wavestatkernel(double *wavedata, double *wavestat,double *wavestat2,byte *hammingArray){

	int keyguess=blockDim.y*blockIdx.y+threadIdx.y;
	int keybyte=blockDim.x*blockIdx.x+threadIdx.x;
	int wave=blockDim.z*blockIdx.z+threadIdx.z;

	if (keyguess<KEYS && keybyte<KEYBYTES && wave<WAVELENGTH ){
		unsigned int i;
		double sigmaWH=0;
		for(i=0;i<SAMPLES;i++){
			sigmaWH+=wavedata[i*WAVELENGTH+wave]*(double)hammingArray[i*KEYS*KEYBYTES + keyguess*KEYBYTES + keybyte];
		}
		wavestat2[wave*KEYS*KEYBYTES + keyguess*KEYBYTES + keybyte ]=sigmaWH;
	}

	if (keyguess==0 && keybyte==0 && wave<WAVELENGTH ){
		unsigned int i;
		double sigmaW=0,sigmaW2=0,W=0;
		for(i=0;i<SAMPLES;i++){
			W=wavedata[i*WAVELENGTH+wave];
			sigmaW+=W;
			sigmaW2+=W*W;
		}
		wavestat[wave]=sigmaW;
		wavestat[WAVELENGTH+wave]=sigmaW2;
	}
	return;
}

//find hamming weight statitics such as signaH sigmaH^2
__global__ void hammingkernel(unsigned int *sample,byte *hammingArray,double *hammingstat){
	int keyguess=blockDim.y*blockIdx.y+threadIdx.y;
	int keybyte=blockDim.x*blockIdx.x+threadIdx.x;

	if (keybyte<KEYBYTES && keyguess<KEYS ){
		double sigmaH=0,sigmaH2=0;
		byte H;
		unsigned int i;
		for(i=0;i<SAMPLES;i++){
			H=hamming(sample,i,keybyte,keyguess);
			hammingArray[i*KEYS*KEYBYTES + keyguess*KEYBYTES + keybyte]=H;
			sigmaH+=(double)H;
			sigmaH2+=(double)H*(double)H;
		}
		hammingstat[KEYBYTES*keyguess+keybyte]=sigmaH;
		hammingstat[KEYS*KEYBYTES+KEYBYTES*keyguess+keybyte]=sigmaH2;
	}
	return;
}


int main(int argc, char *argv[]){
	
	unsigned int i,j;
		
	//check args
	if(argc!=3){
		fprintf(stderr,"%s\n", "Not enough args. eg ./cpa wavedata.txt sample.txt");
		exit(EXIT_FAILURE);
	}
	if(ALLWAVELEN%WAVELENGTH !=0){
		fprintf(stderr,"Make sure that ALLWAVELEN is divisible by WAVELEN\n");
		exit(1);
	}
	
	//allocate RAM for waves
	double *wavedata=(double *)malloc(sizeof(double) * SAMPLES*  WAVELENGTH);
	checkAllocRAM(wavedata);

	//read sample texts
	unsigned int *sample=(unsigned int *)malloc(sizeof(unsigned int)*SAMPLES*KEYBYTES);
	checkAllocRAM(sample);
		
	FILE *file=fopen(argv[2],"r");
	isFileValid(file);
	
	char str[100];
	for(i=0; i<SAMPLES ;i++){
		fscanf(file,"%s",str);
		for(j=0; j<KEYBYTES; j++){
			sscanf(&str[2*j],"%02X",&sample[i*KEYBYTES+j]);		
		}
	}
	fclose(file);


	//space in rAMfor correlation values
	double *corelation=(double *)malloc(sizeof(double) * KEYS * KEYBYTES);
	checkAllocRAM(corelation);
	
	//space for all correlations
	double *allcorelation=(double *)malloc(sizeof(double) * KEYS * KEYBYTES * WAVELENGTH);
	checkAllocRAM(allcorelation);	
	
	//Time
	hipEvent_t start,stop;
	float elapsedtime;
	hipEventCreate(&start);
	hipEventRecord(start,0);
	
	//cuda arrays and copying
	double *dev_wavedata;
	unsigned int *dev_sample;
	double *dev_corelation,*dev_wavestat,*dev_wavestat2,*dev_hammingstat,*dev_allcorrelations;
	byte *dev_hammingArray;
	hipMalloc((void**)&dev_wavedata, SAMPLES*WAVELENGTH*sizeof(double)); 				checkCudaError();
	hipMalloc((void**)&dev_sample, SAMPLES*KEYBYTES*sizeof(unsigned int)); 			checkCudaError();
	hipMalloc((void**)&dev_corelation, KEYS*KEYBYTES*sizeof(double)); 					checkCudaError();
	hipMalloc((void**)&dev_hammingArray, KEYS*KEYBYTES*SAMPLES*sizeof(byte)); 			checkCudaError();
	hipMalloc((void**)&dev_wavestat, 2*WAVELENGTH*sizeof(double)); 					checkCudaError();
	hipMalloc((void**)&dev_wavestat2, KEYS*KEYBYTES*WAVELENGTH*sizeof(double));		checkCudaError();
	hipMalloc((void**)&dev_hammingstat, 2*KEYS*KEYBYTES*sizeof(double)); 				checkCudaError();
	hipMalloc((void**)&dev_allcorrelations, WAVELENGTH*KEYS*KEYBYTES*sizeof(double)); 	checkCudaError();
	
	hipMemset(dev_corelation,0, KEYS*KEYBYTES*sizeof(double)); 						checkCudaError();
	hipMemset(dev_allcorrelations,0, WAVELENGTH*KEYS*KEYBYTES*sizeof(double)); 		checkCudaError();
	hipMemcpy(dev_sample,sample, SAMPLES*KEYBYTES*sizeof(unsigned int),hipMemcpyHostToDevice); checkCudaError();

	//cuda kernel configuraion parameters	
	dim3 grid(KEYBYTES/16,KEYS/16);
	dim3 block(16,16);

	//find hamming statistics
	hammingkernel<<<grid,block>>>(dev_sample,dev_hammingArray,dev_hammingstat);
	hipDeviceSynchronize(); checkCudaError();

	//correlation value writing. Opening files
	char filename[100];
	FILE* filec[KEYBYTES][KEYS];
		
	int keyguess,keybyte;
	for(keybyte=0;keybyte<KEYBYTES;keybyte++){
		for(keyguess=0;keyguess<KEYS;keyguess++){
			sprintf(filename,"subbyte%d_keyguess%d",keybyte,keyguess);
			filec[keybyte][keyguess]=fopen(filename,"w");		
		}
	}	
	
	//start calculations	
	int loops=0;
	for(loops=0;loops<ALLWAVELEN/WAVELENGTH;loops++){

		if(WAVFORMAT==0){
	
			//read wave data
			FILE *file=fopen(argv[1],"r");
			isFileValid(file);
			for(i=0; i<SAMPLES ;i++){
				unsigned int k=0;
				for(j=0; j<ALLWAVELEN; j++){
					float dat;
					fscanf(file,"%f",&dat);
					if(j<WAVELENGTH*(loops+1) && j>=WAVELENGTH*loops){
						wavedata[i*WAVELENGTH+k]=(double)dat;
						k++;
					}
				}
			}	
			fclose(file);
		
		}
		
		else if(WAVFORMAT==1){
			//read wave data
			FILE *file=fopen(argv[1],"rb");
			isFileValid(file);
			for(i=0; i<SAMPLES ;i++){
				fseek(file,sizeof(float)*(i*ALLWAVELEN+WAVELENGTH*loops),SEEK_SET);
				for(j=0; j<WAVELENGTH; j++){
					float dat;
					int ret=fread(&dat,sizeof(float),1,file);
					if(ret<1){
						perror("");
						exit(1);
					}
					wavedata[i*WAVELENGTH+j]=(double)dat;
				}
			}	
			fclose(file);			
			
		}

		else{
			
			fprintf(stderr,"Unknown wave file format\n");
			exit(1);
		}

		//copy wavedata to cuda
		hipMemcpy(dev_wavedata,wavedata,SAMPLES*WAVELENGTH*sizeof(double),hipMemcpyHostToDevice); checkCudaError();
	
		//cuda kernel configuration parameters	
		dim3 block3d(16,16,4);
		dim3 grid3d(KEYBYTES/16,KEYS/16,WAVELENGTH/4);
		
		//find wave statistics
		wavestatkernel<<<grid3d,block3d>>>(dev_wavedata,dev_wavestat,dev_wavestat2,dev_hammingArray);
		hipDeviceSynchronize(); checkCudaError();


		//deploy the correlation calculation and max finding kernel
		maxCorelationkernel<<<grid,block>>>(dev_corelation,dev_wavestat,dev_wavestat2,dev_hammingstat,dev_allcorrelations);
		hipDeviceSynchronize(); checkCudaError();	
		
		//copy intermediate temp valies
		hipMemcpy(allcorelation,dev_allcorrelations,WAVELENGTH*KEYS*KEYBYTES*sizeof(double),hipMemcpyDeviceToHost); checkCudaError();
		
		//write correlation values	
		int point,keyguess,keybyte;
		for(keybyte=0;keybyte<KEYBYTES;keybyte++){
			for(keyguess=0;keyguess<KEYS;keyguess++){		
				for(point=0;point<WAVELENGTH;point++){
						fprintf(filec[keybyte][keyguess],"%f ",allcorelation[point*KEYS*KEYBYTES + keyguess*KEYBYTES + keybyte]);
				}	
				if(loops==ALLWAVELEN/WAVELENGTH-1){
					fprintf(filec[keybyte][keyguess],"\n");				
					fclose(filec[keybyte][keyguess]);
				}
			}
		}		

		//progress
		fprintf(stderr,"%d of %d completed\n",loops+1,ALLWAVELEN/WAVELENGTH);		

	}

	//copy back results
	hipMemcpy(corelation,dev_corelation,KEYS*KEYBYTES*sizeof(double),hipMemcpyDeviceToHost); checkCudaError();
	hipFree(dev_wavedata); 						checkCudaError();
	hipFree(dev_allcorrelations); 					checkCudaError();
	hipFree(dev_sample); 							checkCudaError();
	hipFree(dev_corelation); 						checkCudaError();
	hipFree(dev_wavestat);					 		checkCudaError();
	hipFree(dev_wavestat2); 						checkCudaError();
	hipFree(dev_hammingstat); 						checkCudaError();
	hipFree(dev_hammingArray); 					checkCudaError();
	
	//Time
	hipEventCreate(&stop);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedtime,start,stop);
	fprintf(stderr,"Time spent for CUDA operation : %.10f\n",elapsedtime/(float)1000);

	//form struct array
	struct key_corr key_corrpairs[KEYS][KEYBYTES];
	
	//print all information while putting to structs
	file=fopen(FILEALL,"w");
	for (i=0;i<KEYS;i++){
		for(j=0;j<KEYBYTES;j++){
			key_corrpairs[i][j].key=i;
			key_corrpairs[i][j].corr=corelation[i*KEYBYTES+j];
			fprintf(file,"%.2X : %f\t",i,corelation[i*KEYBYTES+j]);
		}
		fprintf(file,"\n");
	}
	fclose(file);
	
	int k;
	//sort using insertion sort
	for (j=0;j<KEYBYTES;j++){	
		for (i=1;i<KEYS;i++){
			double corr=key_corrpairs[i][j].corr;
			unsigned int key=key_corrpairs[i][j].key;
			for (k=(int)(i-1);k>=0 && corr>key_corrpairs[k][j].corr;k--){
				key_corrpairs[k+1][j].corr=key_corrpairs[k][j].corr;
				key_corrpairs[k+1][j].key=key_corrpairs[k][j].key;
			}
			key_corrpairs[k+1][j].key=key;
			key_corrpairs[k+1][j].corr=corr;
		}
	}
	
	//print all in desceding order
	file=fopen(FILEALLSORTED,"w");
	for (i=0;i<KEYS;i++){
		for(j=0;j<KEYBYTES;j++){
			fprintf(file,"%.2X : %f\t",key_corrpairs[i][j].key,key_corrpairs[i][j].corr);
		}
		fprintf(file,"\n");
	}
	
	//print the best five to  the stdout
	for (i=0;i<5;i++){
		for(j=0;j<KEYBYTES;j++){
			printf("%.2X\t\t\t",key_corrpairs[i][j].key);
		}
		printf("\n");
		for(j=0;j<KEYBYTES;j++){
			printf("%f\t",key_corrpairs[i][j].corr);
		}		
		printf("\n\n");
	}	
	
	return 0;
}



#include "hip/hip_runtime.h"
/*
	
	The A to Z of Building a Testbed for Power Analysis Attacks
	CUDA C source code for Correlation Power Analysis algorithm 
	
	As we tested on this CUDA version (run on NVIDIA Tesla C2075) is more than 1200 faster 
	when compared to a single thread on CPU (Intel i3) 
	and more than 60 times faster when compared to a 32 threaded Xeon processor

	Takes the set of plain text used and the power traces as the inputs.
	Apply Pearson correlation between hypothetical power data calculated through hamming weight
	and real power data in collected power traces.
	Prints the most correlated keys and their respective correlation coefficients
	
	The intermediate value used for the attack is the SBOX output of the first round.
	Hence this is suitable for software AES implementation.
	If you are testing a different intermediate value or a hardware AES implementation change the selection function 
	in the code
	
    Authors : Hasindu Gamaarachchi, Harsha Ganegoda and Roshan Ragel, 
    Department of Computer Engineering, 
    Faculty of Engineering, University of Peradeniya, 22 Dec 2015
 
    For more information read 
    Hasindu Gamaarachchi, Harsha Ganegoda and Roshan Ragel, 
    "The A to Z of Building a Testbed for Power Analysis Attacks", 
    10th IEEE International Conference on Industrial and Information Systems 2015 (ICIIS)]
 
    Any bugs, issues or suggestions please email to hasindu2008@live.com

*/

#include <stdio.h>
#include "helpers.cuh"
#include "data.cuh"


/*************************************************************** CHANGEBLE PARAMETERS ****************************************************************************/
/* Rather that modifying parameters here, it is recommended to run the provided script. The script will take modifiable parameters as arguments 
and will change this source code using "sed" to generate a temporary modified source code called cpatemp.cu
*/


/* file name for all key-correlation pairs sorted in key order
This file after running would contain all the key bytes sorted in key order with their respective correlation coefficients*/
#define FILEALL "all_key_corr.txt"

/* file name for all key-correlation pairs sorted using correlation coefficient
This file after running would contain all the key bytes and their respective correlation coefficients sorted in descending order of the correlation coefficient*/
#define FILEALLSORTED "all_key_corr_sorted.txt"

/* Enable/disable debugging
Comment this out to disable debugging. DIsabling lets kernels and host functions be overlapped*/
//#define DEBUG 1

/************************************************************ END OF CHANGEBLE PARAMETERS ****************************************************************************/


//defined for 128 bit cryptographic algorithm
#define KEYBYTES 16
#define KEYS 256

//struct used for sorting correlation key pairs
struct key_corr{
	unsigned int key;
	double corr;
};


//calculates hamming weight of a 8 bit number
__device__ byte hammingweight(byte H){

	// Count the number of set bits
	byte dist=0;
	while(H){
		dist++; 
		H &= H - 1;
	}
	return dist;
}


/********************************************************************** SELECTION FUNCTION ****************************************************************************/
//This will have to be modified if your selection function/intermediate values are different

//find hamming weight for the selection function
__device__ byte hamming(unsigned int *sample, unsigned int i,unsigned int n,unsigned int key) { //n is byteno  i is the sample
    byte inter = (byte)sbox[sample[i*KEYBYTES+n] ^ key];
    byte dist = hammingweight(inter);	  
	return dist;
}

/********************************************************************END SELECTION FUNCTION ****************************************************************************/


//find the correlation values and then the maximum
__global__ void maxCorelationkernel(double *corelation,double *wavestat,double *wavestat2,double *hammingstat,int SAMPLES,int WAVELENGTH, int EFFECTIVE_WAVELENGTH){
	
	int keyguess=blockDim.y*blockIdx.y+threadIdx.y;
	int keybyte=blockDim.x*blockIdx.x+threadIdx.x;

	if (keybyte<KEYBYTES && keyguess<KEYS ){ 
		
		double sigmaH,sigmaH2,sigmaW=0,sigmaW2=0,sigmaWH=0;	
		sigmaH=hammingstat[KEYBYTES*keyguess+keybyte];
		sigmaH2=hammingstat[KEYS*KEYBYTES+KEYBYTES*keyguess+keybyte];
		double temp_corelation=0;;
		double corelationmax=0;;
		unsigned int j;
		for(j=0;j<EFFECTIVE_WAVELENGTH;j++){
			
			sigmaWH=wavestat2[j*KEYS*KEYBYTES + keyguess*KEYBYTES + keybyte];
			sigmaW=wavestat[j];
			sigmaW2=wavestat[WAVELENGTH+j];

			double numerator=SAMPLES*sigmaWH - sigmaW*sigmaH;
			double denominator=sqrt(SAMPLES*sigmaW2 - sigmaW*sigmaW)*sqrt(SAMPLES*sigmaH2 - sigmaH*sigmaH);
			temp_corelation=fabs(numerator/denominator);
		
			if(temp_corelation>corelationmax){
				corelationmax=temp_corelation;
			}
		}

		if(corelationmax>corelation[keyguess*KEYBYTES+keybyte]){
			corelation[keyguess*KEYBYTES+keybyte]=corelationmax;
		}
	}
	return;
}

//find power trace statistics such as sigmaW sigmaw^2 etc
__global__ void wavestatkernel(double *wavedata, double *wavestat,double *wavestat2,byte *hammingArray,int SAMPLES,int WAVELENGTH, int EFFECTIVE_WAVELENGTH){

	int keyguess=blockDim.y*blockIdx.y+threadIdx.y;
	int keybyte=blockDim.x*blockIdx.x+threadIdx.x;
	int wave=blockDim.z*blockIdx.z+threadIdx.z;

	if (keyguess<KEYS && keybyte<KEYBYTES && wave<EFFECTIVE_WAVELENGTH ){
		unsigned int i;
		double sigmaWH=0;
		for(i=0;i<SAMPLES;i++){
			sigmaWH+=wavedata[i*WAVELENGTH+wave]*(double)hammingArray[i*KEYS*KEYBYTES + keyguess*KEYBYTES + keybyte];
		}
		wavestat2[wave*KEYS*KEYBYTES + keyguess*KEYBYTES + keybyte ]=sigmaWH;
	}

	if (keyguess==0 && keybyte==0 && wave<EFFECTIVE_WAVELENGTH ){
		unsigned int i;
		double sigmaW=0,sigmaW2=0,W=0;
		for(i=0;i<SAMPLES;i++){
			W=wavedata[i*WAVELENGTH+wave];
			sigmaW+=W;
			sigmaW2+=W*W;
		}
		wavestat[wave]=sigmaW;
		wavestat[WAVELENGTH+wave]=sigmaW2;
	}
	return;
}

//find hamming weight statistics such as sigmaH sigmaH^2
__global__ void hammingkernel(unsigned int *sample,byte *hammingArray,double *hammingstat,int SAMPLES){
	int keyguess=blockDim.y*blockIdx.y+threadIdx.y;
	int keybyte=blockDim.x*blockIdx.x+threadIdx.x;

	if (keybyte<KEYBYTES && keyguess<KEYS ){
		double sigmaH=0,sigmaH2=0;
		byte H;
		unsigned int i;
		for(i=0;i<SAMPLES;i++){
			H=hamming(sample,i,keybyte,keyguess);
			hammingArray[i*KEYS*KEYBYTES + keyguess*KEYBYTES + keybyte]=H;
			sigmaH+=(double)H;
			sigmaH2+=(double)H*(double)H;
		}
		hammingstat[KEYBYTES*keyguess+keybyte]=sigmaH;
		hammingstat[KEYS*KEYBYTES+KEYBYTES*keyguess+keybyte]=sigmaH2;
	}
	return;
}


//main function
int main(int argc, char *argv[]){
	
	unsigned int i,j;

	/* The number of power traces used for calculation*/
	int SAMPLES;

	/* The number of sampling points in a single power trace
	This value can be found by inspecting the stat.txt generated after collecting a set of power traces*/
	int ALLWAVELEN;

	/*Global memory on a GPU is limited (RAM as well) and hence if the power traces are large sized, all the things won't fit at once to memory
	If you experience a memory allocation failure when running, reduce WAVELENGTH value
	This will force the program to read the power traces part by part 
	*/
	int WAVELENGTH;

	/* input file format for the power trace file
	0 for ASCII format power traces. 
	1 for binary format. Not readable directly, but file size is less*/
	int WAVFORMAT;
	
	//check args
	if(argc!=5){
		fprintf(stderr,"%s\n", "Not enough args. eg ./cpa <num_of_traces> <sample_points> <power_trace_file> <plain_text_samples_file> ");
		exit(EXIT_FAILURE);
	}
	
	
	//get number of samples
	SAMPLES = atoi(argv[1]);
	//get total wavelength
	ALLWAVELEN = atoi(argv[2]);
	
	//get the waveform filetype
	char *filetype=argv[3]+strlen(argv[3]);
	while(*filetype!='.'){
		filetype--;
	}
	filetype++;
	if(strcmp(filetype,"txt")==0){
		WAVFORMAT=0;
		fprintf(stderr,"File type is ASCII\n");
	}
	else if(strcmp(filetype,"dat")==0){
		WAVFORMAT=1;
		fprintf(stderr,"File type is binary\n");

	}
	else{
		WAVFORMAT=2;
	}

	//check cuda devices
	int nDevices;
	hipGetDeviceCount(&nDevices);
	if(nDevices==0){
		fprintf(stderr,"No CUDA device found. Use the CPU version\n");
		exit(1);
	}
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	fprintf(stderr,"Device name: %s\n", prop.name);
	unsigned long int golabalmem = prop.totalGlobalMem;
	fprintf(stderr,"Total global memory: %lf GB\n",(golabalmem/double(1024*1024*1024)));
	size_t freemem,total;
	hipMemGetInfo(&freemem,&total); 
	fprintf(stderr,"%lf GB free of total %lf GB\n",freemem/double(1024*1024*1024),total/double(1024*1024*1024));

	//fprintf(stderr,"%lf GB free\n",freemem/double(1024*1024*1024));
	
	//required memory in cuda for except wavedata
	unsigned long int size_samples = SAMPLES*KEYBYTES*sizeof(unsigned int);
	unsigned long int size_correlation = KEYS*KEYBYTES*sizeof(double);
	unsigned long int size_hamming = KEYS*KEYBYTES*SAMPLES*sizeof(byte);
	unsigned long int size_hammingstat = 2*KEYS*KEYBYTES*sizeof(double);	
	unsigned long int fixedmem = size_samples+size_correlation+size_hamming+size_hammingstat;
	//fprintf(stderr,"fixed %d\n",fixedmem);
	
	//look for best partition size to match memory requirements.
	//initially start from the largest
	WAVELENGTH = ALLWAVELEN+1000;
	unsigned long int size_waves,size_wavestat,size_wavestat2;
	unsigned long int dynmem;;
	do{
		WAVELENGTH-=1000;
		//required memory for waves
		size_waves = (unsigned long int)SAMPLES*WAVELENGTH*sizeof(double);
		size_wavestat = 2*(unsigned long int)WAVELENGTH*sizeof(double);
		size_wavestat2 = (unsigned long int)KEYS*KEYBYTES*WAVELENGTH*sizeof(double);
		dynmem = size_waves+size_wavestat+size_wavestat2;
		//fprintf(stderr,"%lu %lu %lu\n",size_waves,size_wavestat,size_wavestat2);
	}
	while(fixedmem+dynmem>(unsigned long)freemem);
	//fprintf(stderr,"dynmem %lu\n",dynmem);
	//fprintf(stderr,"addition %lu\n",fixedmem+dynmem);
	//fprintf(stderr,"global %lu\n",golabalmem);
	fprintf(stderr,"Partition size of wavelength is %d\n",WAVELENGTH);	
	
	//number of rounds to  go due to memory limitations 
	int numloops= ceil(ALLWAVELEN/(float)WAVELENGTH);	
	
	//allocate RAM for power traces
	double *wavedata=(double *)malloc(sizeof(double) * SAMPLES*  WAVELENGTH);
	checkAllocRAM(wavedata);

	//allocate RAM for sample texts
	unsigned int *sample=(unsigned int *)malloc(sizeof(unsigned int)*SAMPLES*KEYBYTES);
	checkAllocRAM(sample);
	
	//space in RAM for correlation values
	double *corelation=(double *)malloc(sizeof(double) * KEYS * KEYBYTES);
	checkAllocRAM(corelation);	
	
		
	//open plain text file
	FILE *file=fopen(argv[4],"r");
	isFileValid(file);
	
	//read the plain text samples
	char str[100];
	for(i=0; i<SAMPLES ;i++){
		fscanf(file,"%s",str);
		for(j=0; j<KEYBYTES; j++){
			sscanf(&str[2*j],"%02X",&sample[i*KEYBYTES+j]);		
		}
	}
	fclose(file);


	//start Time measurement
	hipEvent_t start,stop;
	float elapsedtime;
	hipEventCreate(&start);
	hipEventRecord(start,0);
	
	//pointers for GPU
	double *dev_wavedata;
	unsigned int *dev_sample;
	double *dev_corelation,*dev_wavestat,*dev_wavestat2,*dev_hammingstat;
	byte *dev_hammingArray;
	
	//memory allocation in cuda
	hipMalloc((void**)&dev_wavedata, SAMPLES*WAVELENGTH*sizeof(double)); 		checkCudaError();
	hipMalloc((void**)&dev_sample, SAMPLES*KEYBYTES*sizeof(unsigned int));		checkCudaError();
	hipMalloc((void**)&dev_corelation, KEYS*KEYBYTES*sizeof(double));			checkCudaError();
	hipMalloc((void**)&dev_hammingArray, KEYS*KEYBYTES*SAMPLES*sizeof(byte));	checkCudaError();
	hipMalloc((void**)&dev_wavestat, 2*WAVELENGTH*sizeof(double));				checkCudaError();
	hipMalloc((void**)&dev_wavestat2, KEYS*KEYBYTES*WAVELENGTH*sizeof(double));checkCudaError();
	hipMalloc((void**)&dev_hammingstat, 2*KEYS*KEYBYTES*sizeof(double));		checkCudaError();
	
	//make all correlation values 0 at the beginning
	hipMemset(dev_corelation,0, KEYS*KEYBYTES*sizeof(double));		checkCudaError();
	//copy plain text samples to GPU
	hipMemcpy(dev_sample,sample, SAMPLES*KEYBYTES*sizeof(unsigned int),hipMemcpyHostToDevice);	checkCudaError();
	
	//cuda kernel configuraion parameters
	dim3 grid(KEYBYTES/16,KEYS/16);
	dim3 block(16,16);

	//find hamming statistics
	hammingkernel<<<grid,block>>>(dev_sample,dev_hammingArray,dev_hammingstat,SAMPLES);
#ifdef DEBUG		
	hipDeviceSynchronize(); checkCudaError();
#endif
	//start calculations part by part
	int loops=0;
	
	//read power trace data in chunks of WAVELEN
	for(loops=0;loops<numloops;loops++){

		int EFFECTIVE_WAVELENGTH;
		if(ALLWAVELEN%WAVELENGTH!=0 && loops==numloops-1){
			EFFECTIVE_WAVELENGTH = ALLWAVELEN - WAVELENGTH*(numloops-1);
		}
		else{
			EFFECTIVE_WAVELENGTH = WAVELENGTH;
		}	
	
		if(WAVFORMAT==0){
	
			//read wave data
			FILE *file=fopen(argv[3],"r");
			isFileValid(file);
			for(i=0; i<SAMPLES ;i++){
				unsigned int k=0;
				for(j=0; j<ALLWAVELEN; j++){
					float dat;
					fscanf(file,"%f",&dat);
					if(j<WAVELENGTH*(loops+1) && j>=WAVELENGTH*loops){
						wavedata[i*WAVELENGTH+k]=(double)dat;
						k++;
					}
				}
			}	
			fclose(file);
		
		}
		
		else if(WAVFORMAT==1){
			//read wave data
			FILE *file=fopen(argv[3],"rb");
			isFileValid(file);
			for(i=0; i<SAMPLES ;i++){
				fseek(file,sizeof(float)*(i*ALLWAVELEN+WAVELENGTH*loops),SEEK_SET);
				for(j=0; j<EFFECTIVE_WAVELENGTH; j++){
					float dat;
					int ret=fread(&dat,sizeof(float),1,file);
					if(ret<1){
						perror("");
						exit(1);
					}
					wavedata[i*WAVELENGTH+j]=(double)dat;
				}
			}	
			fclose(file);			
			
		}

		else{
			
			fprintf(stderr,"Unknown wave file format\n");
			exit(1);
		}
		
		//copy power tracesto GPU
		hipMemcpy(dev_wavedata,wavedata,SAMPLES*WAVELENGTH*sizeof(double),hipMemcpyHostToDevice); checkCudaError();
		
		//cuda kernel configuration parameters
		dim3 block3d(16,16,4);
		dim3 grid3d(KEYBYTES/16,KEYS/16,WAVELENGTH/4);
		
		//find power trace statistics
		wavestatkernel<<<grid3d,block3d>>>(dev_wavedata,dev_wavestat,dev_wavestat2,
		dev_hammingArray,SAMPLES,WAVELENGTH,EFFECTIVE_WAVELENGTH);
	#ifdef DEBUG
		hipDeviceSynchronize(); checkCudaError();
	#endif
	
		//find the correlation calculation and then the maximums
		maxCorelationkernel<<<grid,block>>>(dev_corelation,dev_wavestat,dev_wavestat2,
		dev_hammingstat,SAMPLES,WAVELENGTH,EFFECTIVE_WAVELENGTH);
	#ifdef DEBUG	
		hipDeviceSynchronize(); checkCudaError();
	#endif	
	
		//print the progress
		fprintf(stderr,"%d of %d being done\n",loops+1,numloops);

	}

	//copy back results from GPU to CPU
	hipMemcpy(corelation,dev_corelation,KEYS*KEYBYTES*sizeof(double),hipMemcpyDeviceToHost);
	
	//free memory in GPU
	hipFree(dev_wavedata);
	hipFree(dev_sample);
	hipFree(dev_corelation);
	hipFree(dev_wavestat);
	hipFree(dev_wavestat2);
	hipFree(dev_hammingstat);
	hipFree(dev_hammingArray);

	//stop Time measurement
	hipEventCreate(&stop);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedtime,start,stop);
	fprintf(stderr,"Time spent for operation : %.10f seconds\n",elapsedtime/(float)1000);

	//form struct array for sorting
	struct key_corr key_corrpairs[KEYS][KEYBYTES];
	
	//print all key-correlation pairs to a file while putting to structs to be later sorted
	file=fopen(FILEALL,"w");
	for (i=0;i<KEYS;i++){
		for(j=0;j<KEYBYTES;j++){
			key_corrpairs[i][j].key=i;
			key_corrpairs[i][j].corr=corelation[i*KEYBYTES+j];
			fprintf(file,"%.2X : %f\t",i,corelation[i*KEYBYTES+j]);
		}
		fprintf(file,"\n");
	}
	fclose(file);
	
	int k;
	//sort based on the correlation coefficient using insertion sort
	for (j=0;j<KEYBYTES;j++){	
		for (i=1;i<KEYS;i++){
			double corr=key_corrpairs[i][j].corr;
			unsigned int key=key_corrpairs[i][j].key;
			for (k=(int)(i-1);k>=0 && corr>key_corrpairs[k][j].corr;k--){
				key_corrpairs[k+1][j].corr=key_corrpairs[k][j].corr;
				key_corrpairs[k+1][j].key=key_corrpairs[k][j].key;
			}
			key_corrpairs[k+1][j].key=key;
			key_corrpairs[k+1][j].corr=corr;
		}
	}
	
	//print all key-correlation pairs sorted in descending order
	file=fopen(FILEALLSORTED,"w");
	for (i=0;i<KEYS;i++){
		for(j=0;j<KEYBYTES;j++){
			fprintf(file,"%.2X : %f\t",key_corrpairs[i][j].key,key_corrpairs[i][j].corr);
		}
		fprintf(file,"\n");
	}
	
	//print the best five correlated keys to the stdout
	for (i=0;i<5;i++){
		for(j=0;j<KEYBYTES;j++){
			printf("%.2X\t\t\t",key_corrpairs[i][j].key);
		}
		printf("\n");
		for(j=0;j<KEYBYTES;j++){
			printf("%f\t",key_corrpairs[i][j].corr);
		}		
		printf("\n\n");
	}	
	
	return 0;
}



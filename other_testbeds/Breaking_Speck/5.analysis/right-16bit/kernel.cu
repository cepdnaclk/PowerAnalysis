#include "hip/hip_runtime.h"
/* Author : Hasindu Gamaarachchi
	CPA for 128/128 bit SPECK software implementation
	To derive right half key K2
	*/
	
#include <stdio.h>
#include "helpers.cuh"
#include "data.cuh"
#include <stdint.h>

//file name for all key-correlation pairs sorted in key order
#define FILEALL "all.txt"
//file name for all key-correlation pairs sorted using correlation coefficient
#define FILEALLSORTED "allsorted.txt"

//set 1 if your samples are hexadecimal separated by spaces
//set 0 if your samples are hexadecimal with no spaces
#define FORMAT 0

//set 0 if need to take fabs(), 1 if negative, 2 if positive
#define CORRELATION_SIGN 0

//Change SAMPLES to the number of power traces
#define SAMPLES 1000
//Change ALLWAVELEGTH to the number of sampling points you have in a single power trace
#define ALLWAVELEN 100000
//Due to memory restrictions on GPU if SAMPLES is large cannot keep all the things at once in memory
//In such case of a memory allocation failure reduce WAVELENGTH
//But make sure that ALLWAVELENGTH is divisible by WAVELENGTH 
#define WAVELENGTH 2000

//define for 128/128 Speck
#define KEYBYTES 16
#define KEYBYTESPART 4
#define KEYS 65536

//struct used for sorting correlation key pairs
struct key_corr{
	unsigned int key;
	double corr;
};


__device__ uint16_t hammingweight(uint16_t H){
	//byte H=M^R;
	// Count the number of set bits
	uint16_t dist=0;
	while(H){
		dist++; 
		H &= H - 1;
	}
	return dist;
}

__device__ uint16_t hamming(unsigned int *sample, unsigned int i,unsigned int n,unsigned int key) { //n is byteno  i is the sample
    
	byte pt0[8];
	copy2(pt0,&sample[i*KEYBYTES]);
	
	byte pt1[8];
	copy2(pt1,&sample[i*KEYBYTES+8]);	
	
	byte ans[8];
	ROR(ans,pt1,8);	
	copy(pt1,ans);		
	_add(ans,pt1,pt0);
	copy(pt1,ans);		
	
	uint16_t answer=(uint16_t)pt1[n*2]<<8|(uint16_t)pt1[n*2+1];
	uint16_t inter;
	if(n<4){	
		 inter= (uint16_t)(answer ^ key);
	}
	else{
		inter = (uint16_t)(answer ^ key);
	}

    uint16_t dist = hammingweight(inter);	  
	return dist;
}


__global__ void maxCorelationkernel(double *corelation,double *wavestat,double *wavestat2,double *hammingstat){
	
	int keyguess=blockDim.y*blockIdx.y+threadIdx.y;
	int keybyte=blockDim.x*blockIdx.x+threadIdx.x;

	if (keybyte<KEYBYTESPART && keyguess<KEYS ){ 
		
		double sigmaH,sigmaH2,sigmaW=0,sigmaW2=0,sigmaWH=0;	
		sigmaH=hammingstat[KEYBYTESPART*keyguess+keybyte];
		sigmaH2=hammingstat[KEYS*KEYBYTESPART+KEYBYTESPART*keyguess+keybyte];
		double temp_corelation=0;;
		double corelationmax=0;;
		unsigned int j;
		for(j=0;j<WAVELENGTH;j++){
			
			sigmaWH=wavestat2[j*KEYS*KEYBYTESPART + keyguess*KEYBYTESPART + keybyte];
			sigmaW=wavestat[j];
			sigmaW2=wavestat[WAVELENGTH+j];

			double numerator=SAMPLES*sigmaWH - sigmaW*sigmaH;
			double denominator=sqrt(SAMPLES*sigmaW2 - sigmaW*sigmaW)*sqrt(SAMPLES*sigmaH2 - sigmaH*sigmaH);

			if(CORRELATION_SIGN==0){
				temp_corelation=fabs(numerator/denominator);
			}
			else if(CORRELATION_SIGN==1){
				temp_corelation=-numerator/denominator;
			}
			else if(CORRELATION_SIGN==2){
				temp_corelation=numerator/denominator;
			}			
			else{
				temp_corelation=fabs(numerator/denominator);
			}			
		
			if(temp_corelation>corelationmax){
				corelationmax=temp_corelation;
			}
		}

		if(corelationmax>corelation[keyguess*KEYBYTESPART+keybyte]){
			corelation[keyguess*KEYBYTESPART+keybyte]=corelationmax;
		}
	}
	return;
}

__global__ void wavestatkernel(double *wavedata, double *wavestat,double *wavestat2,byte *hammingArray){

	int keyguess=blockDim.y*blockIdx.y+threadIdx.y;
	int keybyte=blockDim.x*blockIdx.x+threadIdx.x;
	int wave=blockDim.z*blockIdx.z+threadIdx.z;

	if (keyguess<KEYS && keybyte<KEYBYTESPART && wave<WAVELENGTH ){
		unsigned int i;
		double sigmaWH=0;
		for(i=0;i<SAMPLES;i++){
			sigmaWH+=wavedata[i*WAVELENGTH+wave]*(double)hammingArray[i*KEYS*KEYBYTESPART + keyguess*KEYBYTESPART + keybyte];
		}
		wavestat2[wave*KEYS*KEYBYTESPART + keyguess*KEYBYTESPART + keybyte ]=sigmaWH;
	}

	if (keyguess==0 && keybyte==0 && wave<WAVELENGTH ){
		unsigned int i;
		double sigmaW=0,sigmaW2=0,W=0;
		for(i=0;i<SAMPLES;i++){
			W=wavedata[i*WAVELENGTH+wave];
			sigmaW+=W;
			sigmaW2+=W*W;
		}
		wavestat[wave]=sigmaW;
		wavestat[WAVELENGTH+wave]=sigmaW2;
	}
	return;
}

__global__ void hammingkernel(unsigned int *sample,byte *hammingArray,double *hammingstat){
	int keyguess=blockDim.y*blockIdx.y+threadIdx.y;
	int keybyte=blockDim.x*blockIdx.x+threadIdx.x;

	if (keybyte<KEYBYTESPART && keyguess<KEYS ){
		double sigmaH=0,sigmaH2=0;
		byte H;
		unsigned int i;
		for(i=0;i<SAMPLES;i++){
			H=hamming(sample,i,keybyte,keyguess);
			hammingArray[i*KEYS*KEYBYTESPART + keyguess*KEYBYTESPART + keybyte]=H;
			sigmaH+=(double)H;
			sigmaH2+=(double)H*(double)H;
		}
		hammingstat[KEYBYTESPART*keyguess+keybyte]=sigmaH;
		hammingstat[KEYS*KEYBYTESPART+KEYBYTESPART*keyguess+keybyte]=sigmaH2;
	}
	return;
}


int main(int argc, char *argv[]){
	
	unsigned int i,j;
		
	//check args
	if(argc!=3){
		fprintf(stderr,"%s\n", "Not enough args. eg ./cpa wavedata.txt sample.txt");
		exit(EXIT_FAILURE);
	}
	
	if(ALLWAVELEN%WAVELENGTH !=0){
		fprintf(stderr,"Make sure that ALLWAVELEN is divisible by WAVELEN\n");
		exit(1);
	}	
	
	//get wave data
	double *wavedata=(double *)malloc(sizeof(double) * SAMPLES*  WAVELENGTH);
	isMemoryFull(wavedata);

	//get sample texts
	unsigned int *sample=(unsigned int *)malloc(sizeof(unsigned int)*SAMPLES*KEYBYTES);
	isMemoryFull(sample);
		
FILE *file=fopen(argv[2],"r");
	isFileOK(file);	
	
	if(FORMAT==1){
		for(i=0; i<SAMPLES ;i++){
			for(j=0; j<KEYBYTES; j++){
				fscanf(file,"%x",&sample[i*KEYBYTES+j]);
			}
		}

	}
	
	else if(FORMAT==0){
		char str[100];
		for(i=0; i<SAMPLES ;i++){
			fscanf(file,"%s",str);
			for(j=0; j<KEYBYTES; j++){
				sscanf(&str[2*j],"%02X",&sample[i*KEYBYTES+j]);		
			}
		}
	}
	
	else{
		fprintf(stderr,"Unknown FORMAT for sample text\n");
		exit(1);
	}
	fclose(file);


	//space for corelation
	double *corelation=(double *)malloc(sizeof(double) * KEYS * KEYBYTESPART);
	isMemoryFull(corelation);
	
	//Time
	hipEvent_t start,stop;
	float elapsedtime;
	hipEventCreate(&start);
	hipEventRecord(start,0);
	
	//cuda arrays and copying
	double *dev_wavedata;
	unsigned int *dev_sample;
	double *dev_corelation,*dev_wavestat,*dev_wavestat2,*dev_hammingstat;
	byte *dev_hammingArray;
	checkCudaError(hipMalloc((void**)&dev_wavedata, SAMPLES*WAVELENGTH*sizeof(double)));
	checkCudaError(hipMalloc((void**)&dev_sample, SAMPLES*KEYBYTES*sizeof(unsigned int)));
	checkCudaError(hipMalloc((void**)&dev_corelation, KEYS*KEYBYTESPART*sizeof(double)));
	checkCudaError(hipMalloc((void**)&dev_hammingArray, KEYS*KEYBYTESPART*SAMPLES*sizeof(byte)));
	checkCudaError(hipMalloc((void**)&dev_wavestat, 2*WAVELENGTH*sizeof(double)));
	checkCudaError(hipMalloc((void**)&dev_wavestat2, KEYS*KEYBYTESPART*WAVELENGTH*sizeof(double)));
	checkCudaError(hipMalloc((void**)&dev_hammingstat, 2*KEYS*KEYBYTESPART*sizeof(double)));
	
	checkCudaError(hipMemset(dev_corelation,0, KEYS*KEYBYTESPART*sizeof(double)));
	checkCudaError(hipMemcpy(dev_sample,sample, SAMPLES*KEYBYTES*sizeof(unsigned int),hipMemcpyHostToDevice));
	
	dim3 grid(KEYBYTES/4,KEYS/64);
	dim3 block(4,64);

	//findhamming
	hammingkernel<<<grid,block>>>(dev_sample,dev_hammingArray,dev_hammingstat);
	checkCudaError(hipGetLastError());


	int loops=0;
	for(loops=0;loops<ALLWAVELEN/WAVELENGTH;loops++){

		FILE *file=fopen(argv[1],"r");
		isFileOK(file);
		for(i=0; i<SAMPLES ;i++){
			unsigned int k=0;
			for(j=0; j<ALLWAVELEN; j++){
				float dat;
				fscanf(file,"%f",&dat);
				if(j<WAVELENGTH*(loops+1) && j>=WAVELENGTH*loops){
					wavedata[i*WAVELENGTH+k]=(double)dat;
					k++;
				}
			}
		}	
		fclose(file);

		checkCudaError(hipMemcpy(dev_wavedata,wavedata,SAMPLES*WAVELENGTH*sizeof(double),hipMemcpyHostToDevice));
		
		dim3 block3d(4,32,4);
		dim3 grid3d(KEYBYTESPART/4,KEYS/32,WAVELENGTH/4);
		
		//find wave stats
		wavestatkernel<<<grid3d,block3d>>>(dev_wavedata,dev_wavestat,dev_wavestat2,dev_hammingArray);
		checkCudaError(hipGetLastError());

		//deploy double 
		maxCorelationkernel<<<grid,block>>>(dev_corelation,dev_wavestat,dev_wavestat2,dev_hammingstat);
		checkCudaError(hipGetLastError());	

		//progress
		fprintf(stderr,"%d of %d completed\n",loops+1,ALLWAVELEN/WAVELENGTH);		
		
	}

	//copy back
	checkCudaError(hipMemcpy(corelation,dev_corelation,KEYS*KEYBYTESPART*sizeof(double),hipMemcpyDeviceToHost));
	checkCudaError(hipFree(dev_wavedata));
	checkCudaError(hipFree(dev_sample));
	checkCudaError(hipFree(dev_corelation));
	checkCudaError(hipFree(dev_wavestat));
	checkCudaError(hipFree(dev_wavestat2));
	checkCudaError(hipFree(dev_hammingstat));
	checkCudaError(hipFree(dev_hammingArray));

	//Time
	hipEventCreate(&stop);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedtime,start,stop);
	fprintf(stderr,"Time spent for CUDA operation : %.10f\n",elapsedtime/(float)1000);

	//form struct array
	struct key_corr key_corrpairs[KEYS][KEYBYTESPART];
	
	//print all information while putting to structs	
	file=fopen(FILEALL,"w");
	for (i=0;i<KEYS;i++){
		for(j=0;j<KEYBYTESPART;j++){
			key_corrpairs[i][j].key=i;
			key_corrpairs[i][j].corr=corelation[i*KEYBYTESPART+j];
			fprintf(file,"%.4X : %f\t",i,corelation[i*KEYBYTESPART+j]);
		}
		fprintf(file,"\n");
	}
	
	int k;
	//sort using insertion sort
	for (j=0;j<KEYBYTESPART;j++){	
		for (i=1;i<KEYS;i++){
			double corr=key_corrpairs[i][j].corr;
			unsigned int key=key_corrpairs[i][j].key;
			for (k=(int)(i-1);k>=0 && corr>key_corrpairs[k][j].corr;k--){
				key_corrpairs[k+1][j].corr=key_corrpairs[k][j].corr;
				key_corrpairs[k+1][j].key=key_corrpairs[k][j].key;
			}
			key_corrpairs[k+1][j].key=key;
			key_corrpairs[k+1][j].corr=corr;
		}
	}
	
	//print all in ascending order
	file=fopen(FILEALLSORTED,"w");
	for (i=0;i<KEYS;i++){
		for(j=0;j<KEYBYTESPART;j++){
			fprintf(file,"%.4X : %f\t",key_corrpairs[i][j].key,key_corrpairs[i][j].corr);
		}
		fprintf(file,"\n");
	}
	
	//print the best five to  the stdout
	for (i=0;i<5;i++){
		for(j=0;j<KEYBYTESPART;j++){
			printf("%.4X\t\t",key_corrpairs[i][j].key);
		}
		printf("\n");
		for(j=0;j<KEYBYTESPART;j++){
			printf("%f\t",key_corrpairs[i][j].corr);
		}		
		printf("\n\n");
	}	
		
	return 0;
}


